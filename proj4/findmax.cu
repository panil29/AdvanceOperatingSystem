#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 To compile: 
    nvcc -arch=sm_60 reduction_kernel.cu
 To run with the array size 2^20, expo dist mean 5, and init seed 17:
    ./a.out 20 5 17
 */
#include <stdio.h>
#include <stdlib.h>
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
__global__ void
reduce(double *g_idata, double *g_odata, unsigned int n)
{
    double *sdata = SharedMemory<double>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


// CUDA Runtime
#include <hip/hip_runtime.h>

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
//double findmax(N,M,S);
findmax_gpu(int argc, char **argv)
{
        int n = atoi(argv[1]); // assuming at least 8
        int mean = atoi(argv[2]);
        int seed = atoi(argv[3]); 
        int size = 1<<n;    // number of elements (at least 256) to reduce
        int maxThreads = 256;  // number of threads per block

        // create random input data on CPU
        unsigned int bytes = size * sizeof(double);

        double *h_idata = (double *) malloc(bytes);

        srand48(seed);
        for (int i=0; i<size; i++)
        {
               //h_idata[i] = 1.0; // for testing
               //expo dist with mean 
                h_idata[i] = -mean * log(drand48());
        }

        int numBlocks = size / maxThreads; 
        int numThreads = size;

        int smemSize = maxThreads * sizeof(double);

        // allocate mem for the result on host side
        double *h_odata = (double *) malloc(numBlocks*sizeof(double));

        // allocate device memory and data
        double  *d_idata = NULL;
        double *d_odata = NULL;
        double *d_intermediate = NULL;

        checkCudaErrors(hipMalloc((void **) &d_idata,bytes));
        checkCudaErrors(hipMalloc((void **) &d_odata, numBlocks*sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &d_intermediate, numBlocks*sizeof(double)));

        // copy data directly to device memory
        checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));

        reduce<<<numBlocks,maxThreads,smemSize>>>(d_idata, d_odata, numThreads);

        int s=numBlocks;

        while (s > 1) {
	    checkCudaErrors(hipMemcpy(d_intermediate, d_odata, s * sizeof(double),
                                   hipMemcpyDeviceToDevice));
            reduce<<<(s+maxThreads-1)/maxThreads,maxThreads,smemSize>>>(d_intermediate, d_odata, s);
            s = (s+maxThreads-1)/maxThreads;
        }

        checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(double), hipMemcpyDeviceToHost));

        printf("GPU sum : %f\n\n", h_odata[0]);

        checkCudaErrors(hipFree(d_idata));
        checkCudaErrors(hipFree(d_odata));

    return true;
}
